#include "../inc/duda.h"

#include "../inc/drray.h"

#include "../inc/kernel.h"


#define tx ( threadIdx.x )
#define ty ( threadIdx.y )



st_DeviceArray DeviceArray( uint32_t row, uint32_t col )
{
    st_DeviceArray dary;

    dary.row = row;
    dary.col = col;

    uint32_t size = row * col;

    hipMalloc( &dary.ptr, sizeof( float ) * size );

    hipMemset(  dary.ptr, 0.0, sizeof( float ) * size );

    return dary;
}


st_DeviceArray DeviceIdentity( uint32_t ndim )
{
    st_DeviceArray dary;

    dary.row = ndim;
    dary.col = ndim;

    uint32_t size = ndim * ndim;

    hipMalloc( &dary.ptr, sizeof( float ) * size );

    _Identity_kernel <<<1,ndim>>> ( dary.ptr, dary.row, dary.col );

    hipDeviceSynchronize();

    return dary;
}


error_DUDA Clean( st_DeviceArray* dary )
{
    hipFree( dary->ptr );

    return DUDA_SUCCESS;
}


error_DUDA Print( st_DeviceArray* dary )
{
    uint32_t size = dary->row * dary->col;

    st_Array1D ary1D = Array1D( size );

    hipMemcpy( ary1D.ptr, dary->ptr, sizeof( float ) * size, hipMemcpyDeviceToHost );

    st_Array ary = Array2Array1D( &ary1D, dary->row, dary->col );

    if ( Print( &ary ) != ARR_SUCCESS ) return DUDA_FAILURE;

    Clean( &ary );
    Clean( &ary1D );

    return DUDA_SUCCESS;
}


error_DUDA ElementwiseAdd( st_DeviceArray* OUTPUT, const st_DeviceArray* INPUT1, const st_DeviceArray* INPUT2 )
{
    if ( ( OUTPUT->row != INPUT1->row ) || ( OUTPUT->row != INPUT2->row ) ) return DUDA_UNMATCH;

    if ( ( OUTPUT->col != INPUT1->col ) || ( OUTPUT->col != INPUT2->col ) ) return DUDA_UNMATCH;

    dim3 blockDim( OUTPUT->row, OUTPUT->col );

    _ElementwiseAdd_kernel <<<1, blockDim>>> ( OUTPUT->ptr, INPUT1->ptr, INPUT2->ptr, OUTPUT->row, OUTPUT->col );

    hipDeviceSynchronize();

    return DUDA_SUCCESS;
}


error_DUDA ElementwiseSub( st_DeviceArray* OUTPUT, const st_DeviceArray* INPUT1, const st_DeviceArray* INPUT2 )
{
    if ( ( OUTPUT->row != INPUT1->row ) || ( OUTPUT->row != INPUT2->row ) ) return DUDA_UNMATCH;

    if ( ( OUTPUT->col != INPUT1->col ) || ( OUTPUT->col != INPUT2->col ) ) return DUDA_UNMATCH;

    dim3 blockDim( OUTPUT->row, OUTPUT->col );

    _ElementwiseSub_kernel <<<1, blockDim>>> ( OUTPUT->ptr, INPUT1->ptr, INPUT2->ptr, OUTPUT->row, OUTPUT->col );

    hipDeviceSynchronize();

    return DUDA_SUCCESS;
}


error_DUDA ElementwiseMul( st_DeviceArray* OUTPUT, const st_DeviceArray* INPUT1, const st_DeviceArray* INPUT2 )
{
    if ( ( OUTPUT->row != INPUT1->row ) || ( OUTPUT->row != INPUT2->row ) ) return DUDA_UNMATCH;

    if ( ( OUTPUT->col != INPUT1->col ) || ( OUTPUT->col != INPUT2->col ) ) return DUDA_UNMATCH;

    dim3 blockDim( OUTPUT->row, OUTPUT->col );

    _ElementwiseMul_kernel <<<1, blockDim>>> ( OUTPUT->ptr, INPUT1->ptr, INPUT2->ptr, OUTPUT->row, OUTPUT->col );

    hipDeviceSynchronize();

    return DUDA_SUCCESS;
}


error_DUDA ElementwiseSub( st_DeviceArray* OUTPUT, const st_DeviceArray* INPUT1, const st_DeviceArray* INPUT2 )
{
    if ( ( OUTPUT->row != INPUT1->row ) || ( OUTPUT->row != INPUT2->row ) ) return DUDA_UNMATCH;

    if ( ( OUTPUT->col != INPUT1->col ) || ( OUTPUT->col != INPUT2->col ) ) return DUDA_UNMATCH;

    dim3 blockDim( OUTPUT->row, OUTPUT->col );

    _ElementwiseSub_kernel <<<1, blockDim>>> ( OUTPUT->ptr, INPUT1->ptr, INPUT2->ptr, OUTPUT->row, OUTPUT->col );

    hipDeviceSynchronize();

    return DUDA_SUCCESS;
}
